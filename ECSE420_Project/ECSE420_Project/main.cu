#include<stdio.h>
#include<stdlib.h>
// CUDA runtime
#include<hip/hip_runtime.h>
#include<>

//quick sort
//swap elements
void swap(int* a, int* b) {
	int t = *a;
	*a = *b;
	*b = t;
}

//quick sort
// MODIFICATION: Parallize this part where each thread will access a different element and compare to the pivot rather than do it one at a time
int partition(int arr[], int low, int high)
{
	int pivot = arr[high]; // pivot 
	int i = (low - 1);  // Index of smaller element 

	for (int j = low; j <= high - 1; j++)
	{
		//MODIFICATION: Make each thread go to a specific element and compare rather than one by one
		// If current element is smaller than the pivot 
		if (arr[j] < pivot)
		{
			i++;    // increment index of smaller element 
			swap(&arr[i], &arr[j]);
		}
	}
	swap(&arr[i + 1], &arr[high]);
	return (i + 1);
}
//quick sort
void quickSort(int arr[], int low, int high)
{
	if (low < high)
	{
		/* pi is partitioning index, arr[p] is now
		   at right place */
		int pi = partition(arr, low, high);

		// Separately sort elements before 
		// partition and after partition 
		quickSort(arr, low, pi - 1);
		quickSort(arr, pi + 1, high);
	}
}

//merge sort
//MODIFICATION: Paralleize the merging operation and maybe parallize the splitting operation too
void merge(int arr[], const int l, const int m, const int r)
{
	int i, j, k;
	const int n1 = m - l + 1;
	const int n2 = r - m;

	/* create temp arrays */
	int L[n1], R[n2];

	/* Copy data to temp arrays L[] and R[] */
	for (i = 0; i < n1; i++)
		L[i] = arr[l + i];
	for (j = 0; j < n2; j++)
		R[j] = arr[m + 1 + j];

	/* Merge the temp arrays back into arr[l..r]*/
	i = 0; // Initial index of first subarray 
	j = 0; // Initial index of second subarray 
	k = l; // Initial index of merged subarray 
	while (i < n1 && j < n2)
	{
		if (L[i] <= R[j])
		{
			arr[k] = L[i];
			i++;
		}
		else
		{
			arr[k] = R[j];
			j++;
		}
		k++;
	}

	/* Copy the remaining elements of L[], if there
	   are any */
	while (i < n1)
	{
		arr[k] = L[i];
		i++;
		k++;
	}

	/* Copy the remaining elements of R[], if there
	   are any */
	while (j < n2)
	{
		arr[k] = R[j];
		j++;
		k++;
	}
}

/* l is for left index and r is right index of the
   sub-array of arr to be sorted */
void mergeSort(int arr[], int l, int r)
{
	if (l < r)
	{
		// Same as (l+r)/2, but avoids overflow for 
		// large l and h 
		int m = l + (r - l) / 2;

		// Sort first and second halves 
		mergeSort(arr, l, m);
		mergeSort(arr, m + 1, r);

		merge(arr, l, m, r);
	}
}

void printArray(int A[], int size)
{
	for (int i = 0; i < size; i++)
		printf("%d ", A[i]);
	printf("\n");
}


int main(int args, char* argv[]) {
	//quick sort
	int arr[] = { 10, 7, 8, 9, 1, 5 };
	int n = sizeof(arr) / sizeof(arr[0]);
	quickSort(arr, 0, n - 1);
	printf("Sorted array: n");
	printArray(arr, n);

	//merge sort
	int arr[] = { 12, 11, 13, 5, 6, 7 };
	int arr_size = sizeof(arr) / sizeof(arr[0]);

	printf("Given array is \n");
	printArray(arr, arr_size);

	mergeSort(arr, 0, arr_size - 1);

	printf("\nSorted array is \n");
	printArray(arr, arr_size);
}
