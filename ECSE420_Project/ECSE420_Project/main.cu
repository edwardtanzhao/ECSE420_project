
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
int DEBUG = 1;
char* data;

int* get_list(int len) {

	int* suffix_list = (int*)malloc(len * sizeof(int));
	int i;

	for (i = 0; i < len; i++) {
		suffix_list[i] = i;
	}
	return suffix_list;
}

void quicksort(int* x, int first, int last) {
	int pivot, j, i;
	float temp;

	if (first < last) {
		pivot = first;
		i = first;
		j = last;

		while (i < j) {
			while (x[i] <= x[pivot] && i < last)
				i++;
			while (x[j] > x[pivot])
				j--;
			if (i < j) {
				temp = x[i];
				x[i] = x[j];
				x[j] = temp;
			}
		}

		temp = x[pivot];
		x[pivot] = x[j];
		x[j] = temp;
		quicksort(x, first, j - 1);
		quicksort(x, j + 1, last);

	}
}

void print_suffix_list(int* list, int len) {
	int i = 0;
	for (i = 0; i < len; i++) {
		printf("%d", list[i]);
		if (i != (len - 1)) printf(" ");
	}
	printf("\n");
}

//merge sort
void merge_sort(int i, int j, int a[], int aux[]) {
	if (j <= i) {
		return;     // the subsection is empty or a single element
	}
	int mid = (i + j) / 2;

	// left sub-array is a[i .. mid]
	// right sub-array is a[mid + 1 .. j]

	merge_sort(i, mid, a, aux);     // sort the left sub-array recursively
	merge_sort(mid + 1, j, a, aux);     // sort the right sub-array recursively

	int pointer_left = i;       // pointer_left points to the beginning of the left sub-array
	int pointer_right = mid + 1;        // pointer_right points to the beginning of the right sub-array
	int k;      // k is the loop counter

	// we loop from i to j to fill each element of the final merged array
	for (k = i; k <= j; k++) {
		if (pointer_left == mid + 1) {      // left pointer has reached the limit
			aux[k] = a[pointer_right];
			pointer_right++;
		}
		else if (pointer_right == j + 1) {        // right pointer has reached the limit
			aux[k] = a[pointer_left];
			pointer_left++;
		}
		else if (a[pointer_left] < a[pointer_right]) {        // pointer left points to smaller element
			aux[k] = a[pointer_left];
			pointer_left++;
		}
		else {        // pointer right points to smaller element
			aux[k] = a[pointer_right];
			pointer_right++;
		}
	}

	for (k = i; k <= j; k++) {      // copy the elements from aux[] to a[]
		a[k] = aux[k];
	}
}

int main(int argc, char* argv[]) {
	//quick sort sequential
	/*clock_t start, end;
	double runTime;
	int size = 10;

	start = clock();
	int* data = (int*)malloc((size + 1) * sizeof(int));
	for (int i = 0; i < size; i++) {
		data[i] = i;
	}

	quicksort(data, 0, size-1);
	print_suffix_list(data, size);

	end = clock();
	free(data);

	runTime = (end - start) / (double)CLOCKS_PER_SEC;
	printf("Quicksort sequential size: %d, and runtime: %f\n", size, runTime);*/
	

	clock_t start_m, end_m;
	double runTime_m;
	int size_m = 50;
	int a[100], aux[100], n, i, d, swap;

	for (int i = 0; i < size_m; i++) {
		a[i] = rand() % 50;
	}

	start_m = clock();

	merge_sort(0, size_m - 1, a, aux);

	end_m = clock();

	runTime_m = (end_m - start_m) / (double)CLOCKS_PER_SEC;

	printf("Printing the sorted array:\n");
	for (i = 0; i < size_m; i++)
		printf(" %d, ", a[i]);

	printf("\n");

	printf("Mergesort sequential size: %d, and runtime: %f\n", size_m, runTime_m);
}


